#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

/*
This is the chatGPT solution, in principle it should be trigger with
nvcc -o test_executable test_file.cu -lgtest -lgtest_main -pthread
but `-pthread` is not supported by nvcc. If one removes that bit, one hits
a fatal error gtest/gtest.h: No such file or directory. I installed gtestlib-dev
that seemed to solve the issue.

https://chat.openai.com/c/2bd8791e-753c-441b-9673-61a860643ddd
*/

// Include the CUDA function definition here
__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

// Test fixture class
class CUDATest : public ::testing::Test {
protected:
    // Set up any common resources or configurations needed for the tests
    // For CUDA, you might allocate device memory, etc.
    void SetUp() override {
        // Add setup code here if needed
    }

    // Clean up any resources allocated in SetUp
    void TearDown() override {
        // Add cleanup code here if needed
    }
};

// Test case for the CUDA function
TEST_F(CUDATest, AddFunctionTest) {
    const int arraySize = 5;
    float *x, *y;
    hipMallocManaged(&x, arraySize*sizeof(float));
    hipMallocManaged(&y, arraySize*sizeof(float));

    // Initialize input arrays (you can use random values, etc.)
    for (int i = 0; i < arraySize; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Call the CUDA function
    add<<<1, 1>>>(arraySize, x, y);

    // Copy the result back to the host
    hipDeviceSynchronize();

    // Perform assertions to check if the CUDA function worked as expected
    for (int i = 0; i < arraySize; ++i) {
        EXPECT_EQ(y[i], 3.0f);
    }
}

// Add more test cases as needed

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "../inc/normal_kernel.h"
#include "../inc/arrays.h"

/*
How to run this test suite:

cd cpp/grid-algorithm
make all
./bin/tests
*/

// Test case for the CUDA function
TEST(CUDATest, GenerateRandomVariates) {
    unsigned int numElements = 3;
    hiprandState *devStates;
    float *devResults;
    float mu = 0.0f;
    float sigma = 1.0f;

    // Allocate Memory
    hipMallocManaged(&devStates, numElements * sizeof(hiprandState));
    hipMallocManaged(&devResults, numElements * sizeof(float));

    // Call the CUDA function
    generate_normal_kernel<<<1, 1>>>(
        devStates, numElements, mu, sigma, devResults
    );

    // Copy the result back to the host
    hipDeviceSynchronize();

    // Perform assertions to check if the CUDA function worked as expected
    EXPECT_NEAR(devResults[0], 0.00459315, 1e-5);

    // Free memory
    hipFree(devStates);
    hipFree(devResults);
}


TEST(CUDATest, Linspaces) {
  // SetUp
  const int size = 3;
  const float start = 1.0f;
  const float end = 2.0f;

  float * vecX;
  hipMallocManaged(&vecX, size * sizeof(float));

  // Act
  linspaceCuda(vecX, size, start, end);

  // Assert
  float expectedVecX[size] = {1.0f, 1.5f, 2.0f};
  ASSERT_TRUE(std::equal(vecX, vecX + size, expectedVecX));

  // Tear down
  hipFree(vecX);
}

TEST(CUDATest, GenerateGrids) {
  // SetUp
  const int vecXYSize = 3;
  const int vecZSize = 2;
  const int gridSize = vecXYSize * vecXYSize * vecZSize;

  float *vecX, *vecY, *vecZ;
  float *gridX, *gridY, *gridZ;

  hipMallocManaged(&vecX, vecXYSize * sizeof(float));
  hipMallocManaged(&vecY, vecXYSize * sizeof(float));
  hipMallocManaged(&vecZ, vecZSize * sizeof(float));
  hipMallocManaged(&gridX, gridSize * sizeof(float));
  hipMallocManaged(&gridY, gridSize * sizeof(float));
  hipMallocManaged(&gridZ, gridSize * sizeof(float));

  // Act
  linspaceCuda(vecX, vecXYSize, 1, 3);
  linspaceCuda(vecY, vecXYSize, 1, 3);
  linspaceCuda(vecZ, vecZSize, 4, 5);
  create3dGridCuda(vecX, vecY, vecZ, gridX, gridY, gridZ, vecXYSize, vecZSize);

  // Assert
  float expectedX[18] = {1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 3, 3, 3, 3, 3, 3};
  float expectedY[18] = {1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3};
  float expectedZ[18] = {4, 5, 4, 5, 4, 5, 4, 5, 4, 5, 4, 5, 4, 5, 4, 5, 4, 5};

  ASSERT_TRUE(std::equal(gridX, gridX + gridSize, expectedX));
  ASSERT_TRUE(std::equal(gridY, gridY + gridSize, expectedY));
  ASSERT_TRUE(std::equal(gridZ, gridZ + gridSize, expectedZ));

  // TearDown
  hipFree(vecX);
  hipFree(vecY);
  hipFree(vecZ);
  hipFree(gridX);
  hipFree(gridY);
  hipFree(gridZ);
}

TEST(CUDATest, ComputeLikes) {
  int gridSize = 3;
  float *vecX, *vecY, *vecZ, *likes;

  hipMallocManaged(&vecX, gridSize * sizeof(float));
  hipMallocManaged(&vecY, gridSize * sizeof(float));
  hipMallocManaged(&vecZ, gridSize * sizeof(float));
  hipMallocManaged(&likes, gridSize * sizeof(float));

  linspaceCuda(vecX, gridSize, 20.0f, 22.0f);  // [20, 21, 22]
  linspaceCuda(vecY, gridSize, 1.0f, 3.0f);  // [1, 2, 3]
  linspaceCuda(vecZ, gridSize, 20.0f, 21.0f);  // [20, 20.5, 21]

  computeLikesCuda(likes, vecX, vecY, vecZ, gridSize);

  EXPECT_NEAR(likes[0], 0.39894228, 1e-5);
  EXPECT_NEAR(likes[1], 0.19333406, 1e-5);
  EXPECT_NEAR(likes[2], 0.12579441, 1e-5);

  // TearDown
  hipFree(vecX);
  hipFree(vecY);
  hipFree(vecZ);
  hipFree(likes);
}




int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
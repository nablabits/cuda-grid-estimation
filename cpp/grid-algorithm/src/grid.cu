#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "../inc/cuda_functions.h"
#include "../inc/kernels.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*
Sources:
https://developer.nvidia.com/blog/efficient-cuda-debugging-memory-initialization-and-thread-synchronization-with-nvidia-compute-sanitizer/

https://docs.nvidia.com/cuda/hiprand/group__DEVICE.html

https://chat.openai.com/c/ba212caf-491f-4dfd-ac4f-ce2132672561

TODO: Move them to a note on obsidian once we are done along with the learnings

How to run this:

cd cpp/grid-algorithm
make all

# always check that we are not doing something spooky under the hood
compute-sanintizer ./bin/grid

# Run the binary
./bin/grid
*/

/********************
* Wrapper Functions *
********************/

void computeDensitiesWrapper(float *vecX, float *vecY, float *vecZ, float *output,
                         int startX, int endX, int startY, int endY,
                         int vecXYSize, int vecZSize)
{
  /*
  Wrap the operations needed to compute the densities of the likelihood function.
  */

  /* It seems that we are reinventing the wheel a bit as we could use the
  cuTENSOR library. This, however, has a steeeeep learning curve 😕
  */

  float *gridX, *gridY, *gridZ;
  int gridSize = vecXYSize * vecXYSize * vecZSize;
  hipMallocManaged(&gridX, gridSize * sizeof(float));
  hipMallocManaged(&gridY, gridSize * sizeof(float));
  hipMallocManaged(&gridZ, gridSize * sizeof(float));

  linspaceCuda(vecX, vecXYSize, startX, endX);
  linspaceCuda(vecY, vecXYSize, startY, endY);

  create3dGridCuda(
    vecX, vecY, vecZ, gridX, gridY, gridZ, vecXYSize, vecZSize
  );

  checkArrays(gridX, gridY, gridZ);

  computeDensitiesCuda(output, gridX, gridY, gridZ, gridSize);

  hipFree(gridX);
  hipFree(gridY);
  hipFree(gridZ);
}


void computeLikesWrapper(float *densities, double *likes, int densitiesSize, int likesSize)
{
  /*
  Wrap the operations needed to compute the likelihood function.
  */

  double **likesMatrix;
  int rows = likesSize;  // 101x101 rows
  int cols = densitiesSize / likesSize;  // of 50 elements each

  if (densitiesSize != rows * cols) {
    printf("ERROR: likesSize != rows * cols\n");
    return;
  }

  hipMallocManaged(&likesMatrix, rows * sizeof(double*));
  for (int i = 0; i < rows; i++) {
    hipMallocManaged(&likesMatrix[i], cols * sizeof(double));
  }

  reshapeArray<float, double>(densities, likesMatrix, cols, rows);
  computeLikesCuda(likes, likesMatrix, rows, cols);

  for (int i = 0; i < cols; i++) {
    hipFree(likesMatrix[i]);
  }
  hipFree(likesMatrix);
}


void computeExpectationsWrapper(thrust::device_vector<double> &posterior,
                                int likesSize, float *vectorMu,
                                float *vectorSigma)
{
  /*
  Wrap the operations needed to extract the marginals from the grid and compute
  their expectations

  We have a thrust vector of 10200 elements and we need to convert it to a
  matrix of 101x101 so we can compute the sum over rows (marginal of mu) and the
  sum over columns (marginal of sigma).

  Once we have the marginals we can compute the expected value for each array
  */
  std::cout << "vector first element: " << posterior[0] << std::endl;

  // We start by creating the matrices for the reduction. We need to create two
  // because `marginalize` will modify them to perform the reduction making them
  // not reusable.

  int rows = 101;
  int cols = 101;
  double **posteriorMatrixMu = createMatrix(rows, cols);
  double **posteriorMatrixSigma = createMatrix(rows, cols);

  // Now we reshape the posterior vector. Wemake a host_vector which is more
  // flexible to work with
  thrust::host_vector<double> h_posterior = posterior;

  double *posteriorPtr = h_posterior.data();
  reshapeArray<double, double>(posteriorPtr, posteriorMatrixMu, cols, rows);
  reshapeArray<double, double>(posteriorPtr, posteriorMatrixSigma, cols, rows);

  std::cout << "Matrix first element: " << posteriorMatrixMu[0][0] << std::endl;

  // Define the arrays we will use to store the marginals.
  double *marginalMu, *marginalSigma;
  hipMallocManaged(&marginalMu, cols * sizeof(double));
  hipMallocManaged(&marginalSigma, rows * sizeof(double));

  // Compute the marginals
  marginalizeCuda(marginalMu, posteriorMatrixMu, rows, cols, 1);
  marginalizeCuda(marginalSigma, posteriorMatrixSigma, rows, cols, 0);

  printArrayd(marginalSigma, 5);
  printArrayd(marginalMu, 5);

  // Finally, compute the expectations
  double mu = computeExpectationsCuda(marginalMu, vectorMu, rows);
  double sigma = computeExpectationsCuda(marginalSigma, vectorSigma, rows);

  std::cout << "Inferred mu: " << mu << std::endl;
  std::cout << "Inferred sigma: " << sigma << std::endl;

  // Free up the memory
  freeMatrix(posteriorMatrixMu, rows);
  freeMatrix(posteriorMatrixSigma, rows);
  hipFree(marginalMu);
  hipFree(marginalSigma);
}


int main(void)
{
  /*******************************
  * Generate the Random Variates *
  *******************************/

  /*
  We start by generating the random variates that will serve as a proxy of some
  process in the real world of evidence gathering.
  */

  /* These are the hidden folks we want to estimate*/
  const float mu = 20.0f;
  const float sigma = 2.0f;

  const int rvsSize = 50;
  float *observations;

  CUDA_CALL(hipMallocManaged(&observations, rvsSize * sizeof(float)));

  generateNormalCuda(rvsSize, mu, sigma, observations);

  // Due to seed, the first element should be 18.5689, let's make sure we are
  // we are close enough from it.
  if (observations[0] - 18.5689f > 0.0001f) {
    std::cout << "Oh noh! unexpected observations" << observations[0];
    std::cout << std::endl;
    return 1;
  }

  printArray(observations, rvsSize);

  /*******************
  * Create the Grids *
  *******************/

 /*
  We need a grid with the outer product of three vectors that will represent the
  combinations of the parameters we want to estimate. This is, if our vectors
  are [1, 2, 3] & [4, 5 ,6] & [1, 2] then our outer product will be:

  [1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 3, 3, 3, 3, 3, 3]
  [4, 4, 5, 5, 6, 6, 4, 4, 5, 5, 6, 6, 4, 4, 5, 5, 6, 6]
  [1, 2, 1, 2, 1, 2, 1, 2, 1, 2, 1, 2, 1, 2, 1, 2, 1, 2]

  We use extensively the word `grid`, but it won't be a proper grid as the
  output will be a vector. Granted, a vector with the same elements as the grid,
  v1 x v2 x v3, but a vector after all.
 */

  const int vecSize = 101;
  const int gridSize = vecSize * vecSize * rvsSize;
  const float startMu = 18.0f;
  const float endMu = 22.0f;
  const float startSigma = 1.0f;
  const float endSigma = 3.0f;

  float *vectorMu, *vectorSigma, *densities;

  CUDA_CALL(hipMallocManaged(&vectorMu, vecSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&vectorSigma, vecSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&densities, gridSize * sizeof(float)));
  CUDA_CALL(hipMemset(densities, 0, gridSize * sizeof(int)));

  /*************************
  * Compute the Likelihood *
  *************************/

  /*
  Computing the likelihood involves two steps: First we compute the densities
  over the observations for each pair of mu, sigma. Then, we take the product
  over those densities as they can be thought as a joint probability.

  mus       [   1,    1,    1,    1, ...]  101
  sigmas    [   4,    4,    5,    5, ...]  101
  obs       [   1,    2,    1,    2, ...]  50
  densities [.099, .096, .079, .078, ...]  101x101x50
  likes     [  0.0096,      .0062,   ...]  101x101
  */

  computeDensitiesWrapper(vectorMu, vectorSigma, observations, densities,
                          startMu, endMu, startSigma, endSigma, vecSize,
                          rvsSize);

  double *likes;
  const int likesSize = vecSize * vecSize;
  CUDA_CALL(hipMallocManaged(&likes, likesSize * sizeof(double)));
  computeLikesWrapper(densities, likes, gridSize, likesSize);

  /*************************
   * Compute the Posterior *
  *************************/

  /*
  In principle we will asume a flat prior, which has no impact on the
  likelihoods. But we still need to normalize them so they will add up to 1.
  */


  // We start by building thrust vectors out of the likes array so we can
  // easily and efficiently compute the sum of the array. The first bit is
  // taking the initial value of `likes` and then copying over the rest of the
  // array up to `likesSize` with `likes + likesSize`
  // Then, we just create another vector that will hold the posteriors.

  // TODO: it might be a good idea to use host vectors before and after
  thrust::device_vector<double> likesV(likes, likes + likesSize);
  thrust::device_vector<double> posteriorV(likesSize);
  computePosteriorCuda(likesV, posteriorV, likesSize);

  /*************************
   * Compute the Marginals *
  *************************/

  /*
  Now that we have the posterior we can compute the marginals and with them, the
  expectations for the parameters that hopefully will land closer to the values
  we set to generate the variates.
  */
  computeExpectationsWrapper(posteriorV, likesSize, vectorMu, vectorSigma);


  /**********
  * Cleanup *
  **********/

  // TODO: maybe we can free the elements as soon as we dont need them.
  CUDA_CALL(hipFree(observations));
  CUDA_CALL(hipFree(vectorMu));
  CUDA_CALL(hipFree(vectorSigma));
  CUDA_CALL(hipFree(densities));
  CUDA_CALL(hipFree(likes));

  return 0;
}
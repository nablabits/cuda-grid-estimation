#include <iostream>
#include <hiprand/hiprand_kernel.h>

#include "../inc/normal_kernel.h"
#include "../inc/arrays.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*
Sources:
https://developer.nvidia.com/blog/efficient-cuda-debugging-memory-initialization-and-thread-synchronization-with-nvidia-compute-sanitizer/

https://docs.nvidia.com/cuda/hiprand/group__DEVICE.html

https://chat.openai.com/c/ba212caf-491f-4dfd-ac4f-ce2132672561

TODO: Move them to a note on obsidian once we are done along with the learnings

How to run this:

cd cpp/grid-algorithm
make all

# always check that we are not doing something spooky under the hood
compute-sanintizer ./bin/grid

# Run the binary
./bin/grid
*/


int main(void)
{
  const unsigned int threadsPerBlock = 64;
  const unsigned int blockCount = 64;
  const unsigned int totalThreads = threadsPerBlock * blockCount;  // 4096

  /* These are the hidden folks we want to estimate*/
  const float mu = 20.0f;
  const float sigma = 2.0f;

  /* Generate the random variates */
  /********************************/
  unsigned int numElements = 50;
  hiprandState *devStates;
  float *devResults;

  /* MEMORY ALLOCATION */
  /* Allocate space for prng states */
  CUDA_CALL(hipMallocManaged(&devStates, totalThreads *sizeof(hiprandState)));

  /* Allocate space for results */
  CUDA_CALL(hipMallocManaged(&devResults, totalThreads * sizeof(float)));

  setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

  generate_normal_kernel<<<blockCount, threadsPerBlock>>>(
    devStates, numElements, mu, sigma, devResults
  );

  hipDeviceSynchronize();

  // Due to seed, the first element should be 18.5689, let's check how close
  // we are from it
  if (devResults[0] - 18.5689f > 0.0001f) {
    std::cout << "Oh noh! " << devResults[0] << std::endl;
    return 1;
  }

  /*
  Create the grids
  ****************
  We need a grid with the outer product of two vectors that will represent the
  combinations of the parameters we want to estimate. This is, if our vectors
  are [1, 2, 3] & [4, 5 ,6], then our outer product will be:

  [1, 2, 3, 1, 2, 3, 1, 2, 3]
  [4, 4, 4, 5, 5, 5, 6, 6, 6]

  TODO: we treat the vectors and the grids as separate objects. A possible
  improvement here could be to treat them as a single multidimensional array
  */
  float *vectorMu;
  float *vectorSigma;
  const int size = 101;
  const int gridSize = size * size;
  const float startMu = 18.0f;
  const float endMu = 22.0f;
  const float startSigma = 1.0f;
  const float endSigma = 3.0f;

  float *gridX;
  float *gridY;

  CUDA_CALL(hipMallocManaged(&vectorMu, size * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&vectorSigma, size * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&gridX, gridSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&gridY, gridSize * sizeof(float)));

  createGrid(
    vectorMu, vectorSigma, gridX, gridY, size,
    startMu, endMu, startSigma, endSigma
  );

  // A couple of sanity checks
  if (gridX[5050] != 20.0f) {
    std::cout << "Oh noh! " << gridX[5050] << std::endl;
    return 1;
  }

  // 101 vectorMu * 1/2 vectorSigma; 5050 index
  if (gridY[151] != 2.0f) {
    std::cout << "Oh noh! " << gridY[151] << std::endl;
    return 1;
  }

  /* It may be possible to use thrust::reduce to take the product over axis*/


  /* Cleanup */
  CUDA_CALL(hipFree(devStates));
  CUDA_CALL(hipFree(devResults));
  CUDA_CALL(hipFree(vectorMu));
  CUDA_CALL(hipFree(vectorSigma));
  CUDA_CALL(hipFree(gridX));
  CUDA_CALL(hipFree(gridY));

  return 0;
}
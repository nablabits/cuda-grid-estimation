#include <iostream>
#include <math.h>  // C std library
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "../inc/normal_kernel.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*
Sources:
https://developer.nvidia.com/blog/efficient-cuda-debugging-memory-initialization-and-thread-synchronization-with-nvidia-compute-sanitizer/

https://docs.nvidia.com/cuda/hiprand/group__DEVICE.html

https://chat.openai.com/c/ba212caf-491f-4dfd-ac4f-ce2132672561

TODO: Move them to a note on obsidian once we are done along with the learnings
*/


int main(void)
{
  const unsigned int threadsPerBlock = 64;
  const unsigned int blockCount = 64;

  unsigned int numElements = 50;
  hiprandState *devStates;
  float *devResults;


  /* MEMORY ALLOCATION */
  /* Allocate space for prng states on device */
  CUDA_CALL(hipMallocManaged(&devStates, numElements *sizeof(hiprandState)));

  /* Allocate space for results on device */
  CUDA_CALL(hipMallocManaged(&devResults, numElements * sizeof(float)));

  /* Set results to 0 */
  // CUDA_CALL(
  //   hipMemset(devResults, 0, numElements * sizeof(float))
  // );

  setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

  generate_normal_kernel<<<blockCount, threadsPerBlock>>>(
    devStates, numElements, devResults
  );

  hipDeviceSynchronize();


  unsigned int count = 0;
  unsigned int withinOneSD = 0;
  for (int i = 0; i < numElements; i++) {
    std::cout << devResults[i] << std::endl;
    if (devResults != 0)
      count++;
    if (devResults[i] > -1.0 && devResults[i] < 1.0) {
      withinOneSD++;
    }
  }

  std::cout << "RVs generated: " << count << std::endl;
  std::cout << "Within one SD: " << (float)withinOneSD / count << std::endl;

  /* Cleanup */
  CUDA_CALL(hipFree(devStates));
  CUDA_CALL(hipFree(devResults));

  return 0;
}
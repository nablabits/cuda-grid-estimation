#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "../inc/cuda_functions.h"
#include "../inc/kernels.h"
#include "../inc/wrappers.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

int main(void)
{
  /*******************************
  * Generate the Random Variates *
  *******************************/

  /*
  We start by generating the random variates that will serve as a proxy of some
  process in the real world of evidence gathering.
  */

  /* These are the hidden folks we want to estimate*/
  const float mu = 20.0f;
  const float sigma = 2.0f;

  const int rvsSize = 50;
  float *observations;

  CUDA_CALL(hipMallocManaged(&observations, rvsSize * sizeof(float)));

  generateNormalCuda(rvsSize, mu, sigma, observations);

  // Due to seed, the first element should be 18.5689, let's make sure we are
  // we are close enough from it.
  if (observations[0] - 18.5689f > 0.0001f) {
    std::cout << "Oh noh! unexpected observations" << observations[0];
    std::cout << std::endl;
    return 1;
  }

  /*************************
  * Compute the Likelihood *
  *************************/

  /*
  Computing the likelihood involves two steps: First we compute the densities
  over the observations for each pair of mu, sigma. Then, we take the product
  over those densities as they can be thought as a joint probability.

  mus       [   1,    1,    1,    1, ...]  101
  sigmas    [   4,    4,    5,    5, ...]  101
  obs       [   1,    2,    1,    2, ...]  50
  densities [.099, .096, .079, .078, ...]  101x101x50
  likes     [  0.0096,      .0062,   ...]  101x101
  */

  const int vecSize = 101;
  const int gridSize = vecSize * vecSize * rvsSize;
  const float startMu = mu - 2;
  const float endMu = mu + 2;
  const float startSigma = sigma - 1;
  const float endSigma = sigma + 1;

  float *vectorMu, *vectorSigma, *densities;

  CUDA_CALL(hipMallocManaged(&vectorMu, vecSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&vectorSigma, vecSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&densities, gridSize * sizeof(float)));
  CUDA_CALL(hipMemset(densities, 0, gridSize * sizeof(int)));

  computeDensitiesWrapper(vectorMu, vectorSigma, observations, densities,
                          startMu, endMu, startSigma, endSigma, vecSize,
                          rvsSize);

  CUDA_CALL(hipFree(observations));

  double *likes;
  const int likesSize = vecSize * vecSize;
  CUDA_CALL(hipMallocManaged(&likes, likesSize * sizeof(double)));
  computeLikesWrapper(densities, likes, gridSize, likesSize);

  // We don't need the densities anymore as we now have the likelihoods.
  CUDA_CALL(hipFree(densities));

  /*************************
   * Compute the Posterior *
  *************************/

  /*
  In principle we will asume a flat prior, which has no impact on the
  likelihoods. But we still need to normalize them so they will add up to 1.
  For the normalization we will need to divide each of the values by the sum of
  the whole array.

  We start by building thrust vectors out of the likes array so we can easily
  and efficiently compute the sum of the array. The first bit is taking the
  initial value of `likes` and then copying over the rest of the array up to 
  `likesSize` with `likes + likesSize`. 
  Then, we just create another vector that will hold the posteriors.
  */


  thrust::device_vector<double> likesV(likes, likes + likesSize);
  thrust::device_vector<double> posteriorV(likesSize);
  computePosteriorCuda(likesV, posteriorV);
  CUDA_CALL(hipFree(likes));

  /*************************
   * Compute the Marginals *
  *************************/

  /*
  Now that we have the posterior we can compute the marginals and with them, the
  expectations for the parameters that hopefully will land closer to the values
  we set to generate the variates.
  */
  double* expectations = computeExpectationsWrapper(
    posteriorV, vectorMu, vectorSigma
  );

  std::cout << "Inferred mu: " << expectations[0]
  << "; Actual mu: " << mu <<  std::endl;
  std::cout << "Inferred sigma: " << expectations[1]
  << "; Actual sigma: " << sigma << std::endl;


  /**********
  * Cleanup *
  **********/
  CUDA_CALL(hipFree(vectorMu));
  CUDA_CALL(hipFree(vectorSigma));

  return 0;
}
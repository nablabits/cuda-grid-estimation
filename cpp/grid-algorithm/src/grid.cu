#include <iostream>
#include <hiprand/hiprand_kernel.h>

#include "../inc/normal_kernel.h"
#include "../inc/arrays.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*
Sources:
https://developer.nvidia.com/blog/efficient-cuda-debugging-memory-initialization-and-thread-synchronization-with-nvidia-compute-sanitizer/

https://docs.nvidia.com/cuda/hiprand/group__DEVICE.html

https://chat.openai.com/c/ba212caf-491f-4dfd-ac4f-ce2132672561

TODO: Move them to a note on obsidian once we are done along with the learnings

How to run this:

cd cpp/grid-algorithm
make all

# always check that we are not doing something spooky under the hood
compute-sanintizer ./bin/grid

# Run the binary
./bin/grid
*/


int main(void)
{
  /* These are the hidden folks we want to estimate*/
  const float mu = 20.0f;
  const float sigma = 2.0f;

  /*******************************
  * Generate the random variates *
  *******************************/
  const int rvsSize = 50;
  float *observations;

  CUDA_CALL(hipMallocManaged(&observations, rvsSize * sizeof(float)));

  generateNormalCuda(rvsSize, mu, sigma, observations);

  // Due to seed, the first element should be 18.5689, let's check how close
  // we are from it
  if (observations[0] - 18.5689f > 0.0001f) {
    std::cout << "Oh noh! " << observations[0] << std::endl;
    return 1;
  }

  /*******************
  * Create the grids *
  *******************/

  const int vecSize = 101;
  const int gridSize = vecSize * vecSize * rvsSize;
  const float startMu = 18.0f;
  const float endMu = 22.0f;
  const float startSigma = 1.0f;
  const float endSigma = 3.0f;

  float *vectorMu, *vectorSigma, *likes;

  // TODO: these grid folks are auxiliary constructions only used to compute
  // likes, so we might want to put them in a separate function.
  float *gridX, *gridY, *gridZ;

  CUDA_CALL(hipMallocManaged(&vectorMu, vecSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&vectorSigma, vecSize * sizeof(float)));

  CUDA_CALL(hipMallocManaged(&gridX, gridSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&gridY, gridSize * sizeof(float)));
  CUDA_CALL(hipMallocManaged(&gridZ, gridSize * sizeof(float)));

  linspaceCuda(vectorMu, vecSize, startMu, endMu);
  linspaceCuda(vectorSigma, vecSize, startSigma, endSigma);
  create3dGrid(
    vectorMu, vectorSigma, observations, gridX, gridY, gridZ, vecSize, rvsSize
  );

  checkArrays(gridX, gridY, gridZ);

  /* It seems that we are reinventing the wheel a bit as we could use the
  cuTENSOR library. This, however, has a steeeeep learning curve 😕
  */

  /* Compute the Likelihood Function */
  CUDA_CALL(hipMallocManaged(&likes, gridSize * sizeof(float)));
  CUDA_CALL(hipMemset(likes, 0, gridSize * sizeof(int)));
  computeLikesCuda(likes, gridX, gridY, gridZ, gridSize);

  /**********
  * Cleanup *
  **********/

  CUDA_CALL(hipFree(observations));
  CUDA_CALL(hipFree(vectorMu));
  CUDA_CALL(hipFree(vectorSigma));
  CUDA_CALL(hipFree(gridX));
  CUDA_CALL(hipFree(gridY));
  CUDA_CALL(hipFree(gridZ));
  CUDA_CALL(hipFree(likes));

  return 0;
}
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#include "../inc/normal_kernel.h"
#include "../inc/arrays.h"


// this is a macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

/*
Sources:
https://developer.nvidia.com/blog/efficient-cuda-debugging-memory-initialization-and-thread-synchronization-with-nvidia-compute-sanitizer/

https://docs.nvidia.com/cuda/hiprand/group__DEVICE.html

https://chat.openai.com/c/ba212caf-491f-4dfd-ac4f-ce2132672561

TODO: Move them to a note on obsidian once we are done along with the learnings

How to run this:

cd cpp/grid-algorithm
make all

# always check that we are not doing something spooky under the hood
compute-sanintizer ./bin/grid

# Run the binary
./bin/grid
*/


int main(void)
{
  const unsigned int threadsPerBlock = 64;
  const unsigned int blockCount = 64;
  const unsigned int totalThreads = threadsPerBlock * blockCount;  // 4096

  /* These are the hidden folks we want to estimate*/
  const float mu = 20.0f;
  const float sigma = 5.0f;

  /* Generate the random variates */
  /********************************/
  unsigned int numElements = 50;
  hiprandState *devStates;
  float *devResults;

  /* MEMORY ALLOCATION */
  /* Allocate space for prng states */
  CUDA_CALL(hipMallocManaged(&devStates, totalThreads *sizeof(hiprandState)));

  /* Allocate space for results */
  CUDA_CALL(hipMallocManaged(&devResults, totalThreads * sizeof(float)));

  setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

  generate_normal_kernel<<<blockCount, threadsPerBlock>>>(
    devStates, numElements, mu, sigma, devResults
  );

  hipDeviceSynchronize();

  // Due to seed, the first element should be 16.4222, let's check how close
  // we are from it
  if (devResults[0] - 16.4222f > 0.0001f) {
    std::cout << "Oh noh! " << devResults[0] << std::endl;
    return 1;
  }

  /* 
  Create the grids 
  ****************

  Let's start small with a 3x3 grid, this will mean two arrays of size 9
  in such a way that we can get the outer product of the two 3d-vectors. This
  is, if our 3d vectors are [1, 2, 3] & [4, 5 ,6], then our outer product will
  be:

  [1, 2, 3, 1, 2, 3, 1, 2, 3]
  [4, 4, 4, 5, 5, 5, 6, 6, 6] 
  */

  // TODO: next up, create the linspace equivalent for the arrays and move all
  // to different functions in a separate file

  // we will need the vectors along with the grids for the whole life of the
  // program. Maybe worth initiating them in the beginning of main()
  int vectorX[3] = {1, 2, 3};
  int vectorY[3] = {4, 5, 6};
  const int size = 3;

  int *gridX;
  int *gridY;

  CUDA_CALL(hipMallocManaged(&gridX, totalThreads * sizeof(int)));
  CUDA_CALL(hipMallocManaged(&gridY, totalThreads * sizeof(int)));

  createGrid(vectorX, vectorY, gridX, gridY, size);

  for (int i = 0; i < size * size; ++i) {
      std::cout << gridX[i] << " ";
  }
  std::cout << std::endl;

  for (int i = 0; i < size * size; ++i) {
      std::cout << gridY[i] << " ";
  }
  

  hipFree(gridX);
  hipFree(gridY);


  /* It may be possible to use thrust::reduce to take the product over axis*/


  /* Cleanup */
  CUDA_CALL(hipFree(devStates));
  CUDA_CALL(hipFree(devResults));

  return 0;
}